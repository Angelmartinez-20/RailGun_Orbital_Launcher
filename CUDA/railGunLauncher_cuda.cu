#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>
#include <cmath>
using namespace std;

#define G 9.8

struct launcher {
    int length;
    int angle;
    double stepSize;
    int iterations;
} Launcher;

__constant__ launcher d_launcher;
__device__ double GetVelocity(double a, double b);
__device__ double IntegrateAccel(double time);
__device__ double SIN(double x);

__global__ void computeVelocity(double* velocities) {
    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    if (i <= d_launcher.length) {
        double local_a = (double)(i - 1);
        double local_b = (double)(i);
        velocities[i-1] = GetVelocity(local_a, local_b);
    }
}

int main(int argc, char* argv[]) {
    /* ensures correct input */
    if (argc != 5) {
        cerr << "Usage: " << argv[0] << " <Ramp Length> <Angle> <Step Size> <Sin() Iterations>" << endl;
        exit(-1);
    }

    struct timespec start, end;
    double finalVel = 0.0, altitude, time_taken;
    Launcher.length = atoi(argv[1]);
    Launcher.angle = atoi(argv[2]);
    Launcher.stepSize = atof(argv[3]); 
    Launcher.iterations = atoi(argv[4]); // Corrected to atoi for integer conversion

    /* Copy Launcher struct from host to device */
    hipMemcpyToSymbol(HIP_SYMBOL(d_launcher), &Launcher, sizeof(launcher));

    /* gets device properties */
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    dim3 threadsPerBlock(prop.maxThreadsPerBlock);
    dim3 numBlocks((Launcher.length + threadsPerBlock.x - 1) / threadsPerBlock.x);
    cout << "Number of Blocks: " << numBlocks.x << endl;
    cout << "Threads per Block: " << threadsPerBlock.x << endl;  
      
    clock_gettime(CLOCK_MONOTONIC, &start); 

    /* allocate mem and run on GPU */
    double* d_velocities;
    hipMalloc((void**)&d_velocities, Launcher.length * sizeof(double));
    computeVelocity<<<numBlocks, threadsPerBlock>>>(d_velocities);

    /* allocate mem and compute on CPU */
    double* h_velocities = (double*)malloc(Launcher.length * sizeof(double));
    hipMemcpy(h_velocities, d_velocities, Launcher.length * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < Launcher.length; ++i) {
        finalVel += h_velocities[i];
    }

    hipFree(d_velocities);
    free(h_velocities);

    clock_gettime(CLOCK_MONOTONIC, &end);
    time_taken = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1e9;
    altitude = Launcher.length*sin(Launcher.angle*(M_PI/180));

    printf("Final Velocity: %15.14fm/s, Altitude: %fm, Angled at: %d degrees\n", 
            finalVel, altitude, Launcher.angle);
    printf("Execution time: %f seconds\n", time_taken);

    return 0;
}


/* Implements --> [V0 + sigma A(t)dt] using Runge-Kutta 4 method */
__device__ double GetVelocity(double a, double b) {
    double k1, k2, k3, k4;
    double h = d_launcher.stepSize;
    double y = 0.0;
    int numSteps = (int)((b - a) / h);

    for (int i = 0; i < numSteps; i++) {
        double x0 = a + i * h;
        k1 = h * IntegrateAccel(x0);
        k2 = h * IntegrateAccel(x0 + 0.5 * h);
        k3 = h * IntegrateAccel(x0 + 0.5 * h);
        k4 = h * IntegrateAccel(x0 + h);
        y += (1.0 / 6.0) * (k1 + 2 * k2 + 2 * k3 + k4);
    }

    return y;
}

__device__ double IntegrateAccel(double time) {
    double ascale = 10.236589076381454;                     // increase amplitude of sine curve
    double tscale = d_launcher.length/(M_PI/2);             // gets first 1/4 sine curve
    double flatAccel = SIN(time/tscale)*ascale;             // acceleration when flat ground
    double rampAccel = flatAccel-G*SIN(d_launcher.angle);   // acceleration on ramp (against earth gravity)
    return rampAccel;
}

__device__ double SIN(double x) {
    double term = x, sum = x;
    /* finds sin() with taylor series */
    for (int i = 2; i < d_launcher.iterations*2; i += 2) {
        term = -term * (x*x) / ((double)(i + 1) * i);
        sum += term;
    }
    return sum;
}